
#include "Sobel.h"
#include <assert.h>

#define DEFAULT_THRESHOLD  4000

//#define DEFAULT_FILENAME "BWstop-sign.ppm"
#define DEFAULT_FILENAME "haha.ppm"

void ComputeOnGPU(int* source, int* result, int xsize, int ysize, int thresh);
bool CompareResults(int* result1, int* result2, int ssize, int ysize);

int main(int argc, char **argv)
{
	int thresh = DEFAULT_THRESHOLD;
	char *filename;
	filename = _strdup(DEFAULT_FILENAME);

	if (argc > 1) {
		if (argc == 3)  { // filename AND threshold
			filename = _strdup(argv[1]);
			thresh = atoi(argv[2]);
		}
		if (argc == 2) { // default file but specified threshhold
			thresh = atoi(argv[1]);
		}
		fprintf(stderr, "file %s threshold %d\n", filename, thresh);
	}

	int xsize, ysize, maxval;
	unsigned int *pic = read_ppm(filename, &xsize, &ysize, &maxval);

	int numbytes = xsize * ysize * 3 * sizeof(int);
	int *result_CPU = (int *)malloc(numbytes);
	int *result_GPU = (int *)malloc(numbytes);
	if (!result_CPU || !result_GPU) {
		fprintf(stderr, "sobel() unable to malloc %d bytes\n", numbytes);
		exit(-1); // fail
	}

	int *out_c = result_CPU;
	int *out_g = result_GPU;

	//Init results : all black
	for (int col = 0; col < ysize; col++) {
		for (int row = 0; row < xsize; row++) {
			*out_c++ = 0;
			*out_g++ = 0;
		}
	}

	//the Real Meat
	Sobel_Gold(pic, result_CPU, xsize, ysize, thresh);
	//Compute On Device
	ComputeOnGPU((int*)pic, result_GPU, xsize, ysize, thresh);

	if (!CompareResults(result_CPU, result_CPU, xsize, ysize)){
		write_ppm("result_error.ppm", xsize, ysize, 255, result_GPU);
		fprintf(stderr, "error result, failed\n");
	}

	fprintf(stdout, "sobel success\n");

	write_ppm("result.ppm", xsize, ysize, 255, result_CPU);

	free(result_CPU);
	free(result_GPU);
	free(pic);

	return EXIT_SUCCESS;
}

void ComputeOnGPU(int* source, int* result, int xsize, int ysize, int thresh)
{
	assert(source != NULL && result != NULL);

	int size = xsize * ysize * sizeof(int);
	int *d_source = 0;
	int *d_result = 0;

	hipError_t error;

	error = hipMalloc((void**)&d_source, size);
	error = hipMalloc((void**)&d_result, size);

	if (error != hipSuccess)
	{
		fprintf(stderr, "can not allocate cuda memory\n");
	}

	hipMemcpy(d_source, source, size, hipMemcpyHostToDevice);

	//Kernel Launch




	//End Kernel
	hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

	hipFree(d_source);
	hipFree(d_result);
}

bool CompareResults(int* result1, int* result2, int xsize, int ysize)
{
	//Testing to see if it's correct.
	int error = 0;
	for (int i = 0; i < xsize*ysize; ++i) {
		if (result1[i ] != result2[ i ]) {
			++error;
		}
	}
	if (!error) {
		return true;
	}

	fprintf(stderr, "Difference: %d\n", error);

	return false;
}